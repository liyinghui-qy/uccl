#include "send_cuda.cuh"
#include <iostream>

#define NCCLCHECK(cmd) do {                         \
    ncclResult_t res = cmd;                         \
    if (res != ncclSuccess) {                       \
        std::cerr << "NCCL error: " << ncclGetErrorString(res) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE);                         \
    }                                               \
} while(0)

void nv_gpu_send(void* sendbuff, int count, CCLDatatype datatype, int peer, Communicator* communicator, Stream* stream) {
    ncclDataType_t datatype_cuda = ccl_to_cuda_datatype(datatype);
    ncclComm_t comm = (ncclComm_t) communicator->comm;
    hipStream_t* cudaStream = (hipStream_t*) stream;
    if (stream == nullptr) {
        NCCLCHECK(ncclSend(sendbuff, count, datatype_cuda, peer, comm, 0));
    }
    else {
        NCCLCHECK(ncclSend(sendbuff, count, datatype_cuda, peer, comm, *cudaStream));
    }
    
}