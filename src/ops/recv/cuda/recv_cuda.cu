#include "recv_cuda.cuh"

void nv_gpu_recv(void* recvbuff, int count, CCLDatatype datatype, int peer, Communicator* communicator, Status* status, Stream* stream) {
    ncclDataType_t datatype_cuda = ccl_to_cuda_datatype(datatype);
    ncclComm_t* comm = (ncclComm_t*) communicator;
    hipStream_t* cudaStream = (hipStream_t*) stream;
    ncclRecv(recvbuff, count, datatype_cuda, peer, *comm, *cudaStream);
}
