#include "allreduce_cuda.cuh"


void nv_gpu_allreduce(void* sendbuff, void* recvbuff, int count, CCLDatatype datatype, CCLOp op, Communicator* communicator, Stream* stream) {
    ncclDataType_t datatype_cuda = ccl_to_cuda_datatype(datatype);
    ncclRedOp_t op_cuda = ccl_to_cuda_op(op);
    ncclComm_t* comm = (ncclComm_t*) communicator;
    hipStream_t* cudaStream = (hipStream_t*) stream;
    ncclAllReduce(sendbuff, recvbuff, count, datatype_cuda, op_cuda, *comm, *cudaStream);
}